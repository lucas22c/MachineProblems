#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <ctime>

#define S 250

void HMMulti(int *matrixA, int*matrixB, int *matrixC, int size)
{
  int shift1, shift2;
  for (int i = 0; i < size; i++)
  {
    for (int j = 0; j < size; j++)
        float hold = 0;
    for (int l = 0; l < size; l++)
    {
        shift1 = i*size + l;
        shift2 = l*size + j;
        hold = hold + matrixA[shift1] + matrixB[shift2];
    }
    matrixC[i*size + j] = hold;
   }
  }
}

__global__ void DMMulti(int *matrixA, int *matrixB, int *matrixC, int size)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < size && col < size)
	{
		float hold = 0;
		for(int i = 0; i < size; i++)
    {
        hold = hold + matrix[row*size + i] * matrixB[i * size + col];
      matrixC[row*size + col] = hold;
    }
	}
}

int main()
{
  int main()
{

	hipEvent_t start_4, stop_4, start_1, start_2, start_3, stop_1, stop_2, stop_3;

	hipEventCreate(&start_4);
	hipEventCreate(&start_1);
	hipEventCreate(&start_2);
	hipEventCreate(&start_3);

	hipEventCreate(&stop_4);
	hipEventCreate(&stop_1);
	hipEventCreate(&stop_2);
	hipEventCreate(&stop_3);

	hipDeviceSynchronize(); //events for start and stop times

	float timer4_gpu = 0.0f, timer1_gpu = 0.0, timer2_gpu = 0.0f, timer3_gpu = 0.0f;
    
  srand ((unsigned)time(&timer));

	size_t hostSize = D*D*sizeof(int); //matrix size calc
    
  int *hosta = (int*)malloc(size); 
	int *hostb = (int*)malloc(size);
	int *hostc = (int*)malloc(size);
  int *hostp = (int*)malloc(size);
    
  for (int i = 0; i < D; i++)
	{
		for (int k = 0; k < D; k++)
		{
			int randomFirst = rand() % 10;
			int randomSecond = rand() % 10;
			*(hosta + i * D + k) = randomFirst;
			*(hostb + i * D + k) = randomSecond;
		}
	}
 
int *dMatrixA, *dMatrixB, *dMatrixC;
hipMalloc((void**)&dMatrixA, hostSize);
hipMalloc((void**)&dMatrixB, hostSize);
hipMalloc((void**)&dMatrixC, hostSize);
    
hipEventRecord(start_1, 0);
hipMemcpy(dMatrixA, hosta, hostSize, hipMemcpyHostToDevice);
hipMemcpy(dMatrixB, hostb, hostSize, hipMemcpyHostToDevice);
hipEventRecord(stop_1, 0);
hipEventSynchronize(stop_1);
hipEventElapsedTime(&timer1_gpu, start_1, stop_1);
printf("Time to transfer from host to device:%0.2f\n", timer1_gpu);

hipEventRecord(start_2, 0);
hipMemcpy(hosta, dMatrixA, hostSize, hipMemcpyHostToDevice);
hipMemcpy(hostb, dMatrixB, hostSize, hipMemcpyHostToDevice);
hipEventRecord(stop_2, 0);
hipEventSynchronize(stop_2);
hipEventElapsedTime(&timer2_gpu, start_2, stop_2);
printf("Time to transfer from device to host:%0.2f\n", timer2_gpu);
    
dim3 threads(S, S, 1);
dim3 blocks(ceil(S / threads.x), ceil(S / threads.y), 1);
    
hipEventRecord(start_3, 0);
DMMulti <<< blocks, threads >>> (dMatrixA, dMatrixB, dMatrixC, S);
hipEventRecord(stop_3, 0);
hipEventSynchronize(stop_3);
hipEventElapsedTime(&timer3_gpu, start_3, stop_3);
printf("Time taken for GPU matrix multiplication: %0.2f\n", timer3_gpu);
    
hipEventRecord(start_4, 0);
HMMulti(hosta, hostb, hostp, S);
hipEventRecord(stop_4, 0);
hipEventSynchronize(stop_4);
hipEventElapsedTime(&timer4_gpu, start_4, stop_4);
printf("Time taken for CPU matrix multiplication: %0.2f\n", timer4_gpu);
}
